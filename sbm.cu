#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>

#define TARGET_REPEAT 50 // # of repeats required to stop iterating
#define TIME_STEP 0.1 // a constant that stands for time discretization
#define DETUNING_FREQUENCY 1 // detuning frequency of the Hamiltonian
#define HEAT_PARAMETER 0.06 // heat parameter for the heated algorithm

texture<float, 1, hipReadModeElementType> qubo_tex;
texture<float, 1, hipReadModeElementType> pressure_tex;

/**
 * @brief Set the Pressure object
 * 
 * @param pressure the pressure array to be returned, [0,1)
 * @param dim the size of the array 
 */
__global__ void setPressure(float* pressure, int dim) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < dim) {
        pressure[id] = id * DETUNING_FREQUENCY / (float)dim;
    }
}

/**
 * @brief calculate the dot product of given two arrays
 * 
 * @param product the product to be returned
 * @param spin the spin array
 * @param dim the dimention of the array
 */
__global__ void dot(float* product, float* spin, int dim) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < dim) {
        product[id] = 0;
        for (int i = 0; i < dim; i++) {
            product[id] += tex1Dfetch(qubo_tex, id * dim + i) * (spin[i] > 0 ? 1 : (spin[i] < 0 ? -1 : 0));
        }
    }
}

/**
 * @brief create an array of random numbers between (-1,1)
 * 
 * @param arr the array to be returned
 * @param size the size of the array
 */
__global__ void initRand(float* arr, int size) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < size) {
        hiprandState state;
        hiprand_init(clock64(), id, 0, &state);
        arr[id] = 2 * hiprand_uniform(&state) - 1;
    }
}

/**
 * @brief calculate the standard deviation of given array
 *
 * @param arr input array
 * @param size size of the array
 * @return the standard deviation of the array
 */
float stddiv(float* arr, int size) {
    float mean = 0;
    for (int i = 0; i < size; i++) {
        mean += arr[i];
    }
    mean /= size;
    float sumDistanceSquare = 0;
    for (int i = 0; i < size; i++) {
        sumDistanceSquare += pow(arr[i] - mean, 2);
    }
    return sqrt(sumDistanceSquare / size);
}

/**
 * @brief update the spin and its momemtum
 *
 * @param spin the spin array
 * @param momentum momentum of the spin
 * @param qubo the relationship matrix parsed to 1D
 * @param dim size of the array
 * @param step the # of step
 * @param xi0 a constant calculated with qubo matrix
 */
__global__ void update(float* spin, float* momentum, float* dot_product, int dim, int step, float xi0) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < dim) {
        momentum[id] += TIME_STEP * ((tex1Dfetch(pressure_tex, step) - DETUNING_FREQUENCY) * spin[id] + xi0 * dot_product[id]);
        spin[id] += TIME_STEP * DETUNING_FREQUENCY * momentum[id];
    }
}

/**
 * @brief bound spins within the range [-1,1]
 *
 * @param spin the spin array
 * @param momentum the momentum of the array
 * @param dim size of the spin array
 */
__global__ void confine(float* spin, float* momentum, int dim) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < dim) {
        if (spin[id] < -1) {
            spin[id] = -1;
            momentum[id] = 0;
        } else if (spin[id] > 1) {
            spin[id] = 1;
            momentum[id] = 0;
        }
    }
}

/**
 * @brief calculate the heated momentum of the spins
 *
 * @param momentum the momentum of the spin
 * @param pastMomentum the previous momentum of the spin
 * @param dim the size of the spin array
 */
__global__ void heatUp(float* momentum, float* pastMomentum, int dim) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < dim) {
        momentum[id] += pastMomentum[id] * TIME_STEP * HEAT_PARAMETER;
    }
}

/**
 * @brief check if the given two spin array have same spin
 *
 * @param spin1 first spin array
 * @param spin2 second spin array
 * @param dim size of spin array
 * @return 0 if true, else false
 */
int sameSpin(float* spin1, float* spin2, int dim) {
    int sameCount = 0;
    for (int i = 0; i < dim; i++) {
        sameCount += spin1[i] * spin2[i] > 0 ? 1 : 0;
    }
    // printf("--not same count: %d--\n", dim - sameCount);
    return sameCount;
}

extern "C" {
    void iterate(float* spin, float* qubo, int dim, int window, int maxStep);
}

/**
 * @brief the iteration step of the simulated bifurcation algorithm
 *
 * @param spin the spin array
 * @param qubo the relationship matrix pased into 1D
 * @param dim dimention of the spin array
 * @param window number of time steps between two spin sampling. if 0 then no window used
 * @param maxStep maximum iteration of the algorithm
 */
void iterate(float* spin, float* qubo, int dim, int window, int maxStep) {

    if (window < 0) {
        printf("window cannot be negative.\n");
        exit(-1);
    }

    int device;
    hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);

    int blocks = 32 * 16;
    int threads = dim / blocks + 1;
    while (threads > prop.maxThreadsPerBlock) {
        blocks *= 2;
        threads = dim / blocks + 1;
    }

    float* momentum;
    hipMalloc(&momentum, dim * sizeof(float));
    initRand << <blocks, threads >> > (momentum, dim);
    float* pastMomentum;
    hipMalloc(&pastMomentum, dim * sizeof(float));
    hipDeviceSynchronize();

    float xi0;
    xi0 = (0.7 * DETUNING_FREQUENCY) / stddiv(qubo, dim * dim) * sqrt(dim);

    float** sample;
    if (window != 0) {
        sample = (float**)malloc((maxStep / window + 1) * sizeof(float*));
        for (int i = 0; i < maxStep / window + 1; i++) {
            sample[i] = (float*)malloc(dim * sizeof(float));
        }
    }

    float* dot_product;
    hipMalloc(&dot_product, dim * sizeof(float));

    float* spin_dev;
    hipMalloc(&spin_dev, dim * sizeof(float));
    hipMemcpy(spin_dev, spin, dim * sizeof(float), hipMemcpyHostToDevice);

    float* qubo_dev;
    hipMalloc(&qubo_dev, dim * dim * sizeof(float));
    hipMemcpy(qubo_dev, qubo, dim * dim * sizeof(float), hipMemcpyHostToDevice);
    hipBindTexture(0, qubo_tex, qubo_dev, dim * dim * sizeof(float));

    float* pressure;
    hipMalloc(&pressure, dim * sizeof(float));
    setPressure << <50, threads >> > (pressure, dim);
    hipBindTexture(0, pressure_tex, pressure, dim * sizeof(float));

    if (window == 0) {
        for (int i = 0; i < maxStep; i++) {
            hipMemcpy(pastMomentum, momentum, dim * sizeof(float), hipMemcpyDeviceToDevice);
            dot << <blocks, threads >> > (dot_product, spin_dev, dim);
            // hipDeviceSynchronize();
            update << <blocks, threads >> > (spin_dev, momentum, dot_product, dim, i, xi0);
            //hipDeviceSynchronize();
            confine << <blocks, threads >> > (spin_dev, momentum, dim);
            //hipDeviceSynchronize();
            heatUp << <blocks, threads >> > (momentum, pastMomentum, dim);
            hipDeviceSynchronize();
        }
    } else {
        int repeatNum = 0;
        for (int i = 0; i < maxStep; i++) {
            hipMemcpy(pastMomentum, momentum, dim * sizeof(float), hipMemcpyDeviceToDevice);
            dot << <blocks, threads >> > (dot_product, spin_dev, dim);
            // hipDeviceSynchronize();
            update << <blocks, threads >> > (spin_dev, momentum, dot_product, dim, i, xi0);
            // hipDeviceSynchronize();
            confine << <blocks, threads >> > (spin_dev, momentum, dim);
            // hipDeviceSynchronize();
            heatUp << <blocks, threads >> > (momentum, pastMomentum, dim);
            hipDeviceSynchronize();
            if (i % window == 0) {
                hipMemcpy(sample[i / window], spin_dev, dim * sizeof(float), hipMemcpyDeviceToHost);
                if (i != 0) {
                    sameSpin(sample[i / window], sample[i / window - 1], dim) == dim ? (repeatNum++) : (repeatNum = 0);
                    if (repeatNum == TARGET_REPEAT) {
                        printf("meet criteria at step = %d\n", i);
                        break;
                    }
                }
            }
        }
    }

    hipMemcpy(spin, spin_dev, dim * sizeof(float), hipMemcpyDeviceToHost);

    hipUnbindTexture(&qubo_tex);
    hipUnbindTexture(&pressure_tex);

    if (window != 0) {
        for (int i = 0; i < maxStep / window + 1; i++) {
            free(sample[i]);
        }
        free(sample);
    }
    hipFree(spin_dev);
    hipFree(qubo_dev);
    hipFree(dot_product);
    hipFree(momentum);
    hipFree(pastMomentum);
    hipFree(pressure);
}

int main() {

    float spin[] = { 0.0f,0.0f };
    float qubo[] = { 0.0f, 1.0f, 1.0f, 0.0f };
    int dim = 2;
    int window = 0;
    int maxStep = 200;
    iterate(spin, qubo, dim, window, maxStep);

    for (int i = 0; i < dim; i++) {
        printf("%s", spin[i] > 0 ? "+" : "-");
    }
    printf("\n");
}