#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>

#define TARGET_REPEAT 35 // # of repeats required to stop iterating
#define TIME_STEP 0.01 // a constant that stands for time discretization
#define PRESSURE_SLOPE 0.01 // pumping pressure's linear slope allowing adiabatic evolution
#define DETUNING_FREQUENCY 1 // detuning frequency of the Hamiltonian
#define HEAT_PARAMETER 0.06 // heat parameter for the heated algorithm

__global__ void dot(float* product, float* qubo, float* spin, int dim) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < dim) {
        product[id] = 0;
        for (int i = 0; i < dim; i++) {
            product[id] += qubo[id * dim + i] * (spin[i] > 0 ? 1 : (spin[i] < 0 ? -1 : 0));
        }
    }
}

__global__ void initRand(float* arr, int size) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < size) {
        hiprandState state;
        hiprand_init(clock64(), id, 0, &state);
        arr[id] = 2 * hiprand_uniform(&state) - 1;
    }
}

/**
 * @brief calculate the standard deviation of given array
 *
 * @param arr input array
 * @param size size of the array
 * @return the standard deviation of the array
 */
float stddiv(float* arr, int size) {
    float mean = 0;
    for (int i = 0; i < size; i++) {
        mean += arr[i];
    }
    mean /= size;
    float sumDistanceSquare = 0;
    for (int i = 0; i < size; i++) {
        sumDistanceSquare += pow(arr[i] - mean, 2);
    }
    return sqrt(sumDistanceSquare / size);
}

/**
 * @brief update the spin and its momemtum
 *
 * @param spin the spin array
 * @param momentum momentum of the spin
 * @param qubo the relationship matrix parsed to 1D
 * @param dim size of the array
 * @param step the # of step
 * @param xi0 a constant calculated with qubo matrix
 */
__global__ void update(float* spin, float* momentum, float* dot_product, int dim, int step, float xi0) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < dim) {
        float pressure = PRESSURE_SLOPE * TIME_STEP * step;
        momentum[id] += TIME_STEP * ((pressure - DETUNING_FREQUENCY) * spin[id] + xi0 * dot_product[id]);
        spin[id] += TIME_STEP * DETUNING_FREQUENCY * momentum[id];
    }
    // float pressure = PRESSURE_SLOPE * TIME_STEP * step;

    // for (int i = 0; i < dim; i++) {
    //     float dot_product = 0;
    //     for (int j = 0; j < dim; j++) {
    //         dot_product += qubo[i * dim + j] * (spin[j] > 0 ? 1 : (spin[j] < 0 ? -1 : 0));
    //     }
    //     momentum[i] += TIME_STEP * ((pressure - DETUNING_FREQUENCY) * spin[i] + xi0 * dot_product);
    //     spin[i] += TIME_STEP * DETUNING_FREQUENCY * momentum[i];
    //     // printf("%s", spin[i] > 0 ? "+" : "-");
    // }
    // // printf("\n");


}

/**
 * @brief bound spins within the range [-1,1]
 *
 * @param spin the spin array
 * @param momentum the momentum of the array
 * @param dim size of the spin array
 */
__global__ void confine(float* spin, float* momentum, int dim) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < dim) {
        if (spin[id] < -1) {
            spin[id] = -1;
            momentum[id] = 0;
        } else if (spin[id] > 1) {
            spin[id] = 1;
            momentum[id] = 0;
        }
    }
}

/**
 * @brief calculate the heated momentum of the spins
 *
 * @param momentum the momentum of the spin
 * @param pastMomentum the previous momentum of the spin
 * @param dim the size of the spin array
 */
__global__ void heatUp(float* momentum, float* pastMomentum, int dim) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < dim) {
        momentum[id] += pastMomentum[id] * TIME_STEP * HEAT_PARAMETER;
    }
}

/**
 * @brief check if the given two spin array have same spin
 *
 * @param spin1 first spin array
 * @param spin2 second spin array
 * @param dim size of spin array
 * @return 0 if true, else false
 */
int sameSpin(float* spin1, float* spin2, int dim) {
    int sameCount = 0;
    for (int i = 0; i < dim; i++) {
        sameCount += spin1[i] * spin2[i] > 0 ? 1 : 0;
    }
    // printf("--not same count: %d--", dim - sameCount);
    return dim - sameCount;
}

extern "C" {
    void iterate(float* spin, float* qubo, int dim, int window, int maxStep);
}

/**
 * @brief the iteration step of the simulated bifurcation algorithm
 *
 * @param spin the spin array
 * @param qubo the relationship matrix pased into 1D
 * @param dim dimention of the spin array
 * @param window number of time steps between two spin sampling. if 0 then no window used
 * @param maxStep maximum iteration of the algorithm
 */
void iterate(float* spin, float* qubo, int dim, int window, int maxStep) {

    if (window < 0) {
        printf("window cannot be negative.\n");
        exit(-1);
    }

    float* momentum;
    hipMalloc(&momentum, dim * sizeof(float));
    initRand << <50, 50 >> > (momentum, dim);
    float* pastMomentum;
    hipMalloc(&pastMomentum, dim * sizeof(float));
    hipDeviceSynchronize();

    float xi0;
    xi0 = (0.7 * DETUNING_FREQUENCY) / stddiv(qubo, dim * dim) * sqrt(dim);

    float** sample;
    if (window != 0) {
        sample = (float**)malloc((maxStep / window + 1) * sizeof(float*));
        for (int i = 0; i < maxStep / window + 1; i++) {
            sample[i] = (float*)malloc(dim * sizeof(float));
        }
    }

    float* dot_product;
    hipMalloc(&dot_product, dim * sizeof(float));

    float* spin_dev;
    hipMalloc(&spin_dev, dim * sizeof(float));
    hipMemcpy(spin_dev, spin, dim * sizeof(float), hipMemcpyHostToDevice);

    float* qubo_dev;
    hipMalloc(&qubo_dev, dim * dim * sizeof(float));
    hipMemcpy(qubo_dev, qubo, dim * dim * sizeof(float), hipMemcpyHostToDevice);

    if (window == 0) {
        for (int i = 0; i < maxStep; i++) {
            hipMemcpy(pastMomentum, momentum, dim * sizeof(float), hipMemcpyDeviceToDevice);
            dot << <50, 50 >> > (dot_product, qubo_dev, spin_dev, dim);
            hipDeviceSynchronize();
            update << <50, 50 >> > (spin_dev, momentum, dot_product, dim, i, xi0);
            //hipDeviceSynchronize();
            confine << <50, 50 >> > (spin_dev, momentum, dim);
            //hipDeviceSynchronize();
            heatUp << <50, 50 >> > (momentum, pastMomentum, dim);
            hipDeviceSynchronize();
        }
    } else {
        int repeatNum = 0;
        for (int i = 0; i < maxStep; i++) {
            hipMemcpy(pastMomentum, momentum, dim * sizeof(float), hipMemcpyDeviceToDevice);
            dot << <50, 50 >> > (dot_product, qubo_dev, spin_dev, dim);
            hipDeviceSynchronize();
            update << <50, 50 >> > (spin_dev, momentum, dot_product, dim, i, xi0);
            // hipDeviceSynchronize();
            confine << <50, 50 >> > (spin_dev, momentum, dim);
            // hipDeviceSynchronize();
            heatUp << <50, 50 >> > (momentum, pastMomentum, dim);
            if (i % window == 0) {
                hipMemcpy(sample[i / window], spin_dev, dim * sizeof(float), hipMemcpyDeviceToHost);
                if (i != 0) {
                    sameSpin(sample[i / window], sample[i / window - 1], dim) == 0 ? (repeatNum++) : (repeatNum = 0);
                    if (repeatNum == TARGET_REPEAT) {
                        // printf("meet criteria at step = %d\n", i);
                        break;
                    }
                }
            }
            hipDeviceSynchronize();
        }
    }

    hipMemcpy(spin, spin_dev, dim * sizeof(float), hipMemcpyDeviceToHost);

    if (window != 0) {
        for (int i = 0; i < maxStep / window + 1; i++) {
            free(sample[i]);
        }
        free(sample);
    }
    hipFree(spin_dev);
    hipFree(qubo_dev);
    hipFree(dot_product);
    hipFree(momentum);
    hipFree(pastMomentum);
}

int main() {

    float spin[] = { 0.5f,-0.5f };
    float qubo[] = { 0.0f, 1.0f, 1.0f, 0.0f };
    int dim = 2;
    int window = 0;
    int maxStep = 200;
    iterate(spin, qubo, dim, window, maxStep);

    for (int i = 0; i < dim; i++) {
        printf("%s", spin[i] > 0 ? "+" : "-");
    }
    printf("\n");
}